#include "hip/hip_runtime.h"
#include "header.h"

int toTest = 1;
int kernPar1 = 6;
int kernPar2 = 256;

const int xRes = 1600, yRes = 900;

void MainWindow() {
	// initiation of window
	ySize = xSize * yRes / xRes;
	srand((unsigned int)time(NULL));
	SDL_Init(SDL_INIT_VIDEO);
	sdl_window = SDL_CreateWindow("MyWindow", 50, 30, xRes, yRes, SDL_WINDOW_RESIZABLE);
	if (sdl_window == NULL) {
		printf(" > unable to open sdl_window\n");
		exit(EXIT_FAILURE);
	}
	sdl_renderer = SDL_CreateRenderer(sdl_window, -1, SDL_RENDERER_ACCELERATED);
	if (sdl_renderer == NULL) {
		printf(" > unable to open sdl_renderer\n");
		exit(EXIT_FAILURE);
	}
	SDL_SetRenderDrawColor(sdl_renderer, 255, 0, 0, 255);
	SDL_SetRenderDrawBlendMode(sdl_renderer, SDL_BLENDMODE_NONE);
	sdl_texture = SDL_CreateTexture(sdl_renderer,
		SDL_PIXELFORMAT_ARGB8888, SDL_TEXTUREACCESS_STATIC, xRes, yRes);
	if (sdl_texture == NULL) {
		printf(" > unable to open sdl_texture\n");
		exit(EXIT_FAILURE);
	}

	hipMallocManaged(&pixels, sizeof(Uint32) * xRes * yRes);
	hipDeviceSynchronize();

	// working loop
	SDL_Event event;
	int beginTime = 0, timeTaken = 0;
	int upd = 1, quit = 0;

	while (!quit)
	{
		beginTime = SDL_GetTicks();
		while (SDL_PollEvent(&event) == 1) {
			if (event.type == SDL_QUIT) {
				quit = 1;
			}
			else {
				switch (event.type) {
				case SDL_KEYDOWN:
					upd = 1;
					switch (event.key.keysym.sym) {
					case SDLK_w:
					case SDLK_UP:
						yPos -= ySize / 10;
						break;
					case SDLK_s:
					case SDLK_DOWN:
						yPos += ySize / 10;
						break;
					case SDLK_d:
					case SDLK_RIGHT:
						xPos += xSize / 10;
						break;
					case SDLK_a:
					case SDLK_LEFT:
						xPos -= xSize / 10;
						break;
					case SDLK_q:
						xPos -= xSize * (1 - 1 / 1.25f) / 2;
						yPos -= ySize * (1 - 1 / 1.25f) / 2;
						xSize *= 1.25; ySize *= 1.25;
						break;
					case SDLK_e:
						xPos += xSize * (1 - 1 / 1.25f) / 2;
						yPos += ySize * (1 - 1 / 1.25f) / 2;
						xSize /= 1.25; ySize /= 1.25;
						break;
					default:
						break;
					}
					break;
				case SDL_MOUSEWHEEL:
					upd = 1;
					if (event.wheel.y > 0) {
						xPos += xSize * (1 - 1 / 1.25f) / 2;
						yPos += ySize * (1 - 1 / 1.25f) / 2;
						xSize /= 1.25; ySize /= 1.25;
					}
					else if (event.wheel.y < 0) {
						xPos -= xSize * (1 - 1 / 1.25f) / 2;
						yPos -= ySize * (1 - 1 / 1.25f) / 2;
						xSize *= 1.25; ySize *= 1.25;
					}
					break;
				}
			}
		}

		upd |= toTest;
		if (upd == 1) {
			update_pixels();
			upd = 0;
		}

		// update window
		SDL_UpdateTexture(sdl_texture, NULL, pixels, xRes * sizeof(Uint32));
		SDL_RenderCopy(sdl_renderer, sdl_texture, NULL, NULL);
		SDL_RenderPresent(sdl_renderer);

		int frameTime = 15;
		timeTaken = SDL_GetTicks() - beginTime;
		if (timeTaken < frameTime) {
			SDL_Delay(frameTime - timeTaken);
		}
		else {
			printf("Time of frame: %d\n", timeTaken);
		}
	}

	// close window
	hipDeviceSynchronize();
	hipFree(pixels);
	SDL_DestroyTexture(sdl_texture);
	SDL_DestroyWindow(sdl_window);
	SDL_Quit();
}

void update_pixels() {
	int* res = NULL;
	float* pos = NULL;

	hipMallocManaged(&res, sizeof(int) * 2);
	hipMallocManaged(&pos, sizeof(float) * 2);

	res[0] = xRes; res[1] = yRes;

	pos[0] = xPos; pos[1] = yPos;
	pos[2] = xSize; pos[3] = ySize;

	kernel <<< kernPar1, kernPar2 >>> (pixels, res, pos);
	hipFree(res);
	hipFree(pos);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stdout, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

__global__ void kernel(Uint32* pixels, int* res, float* pos) {
	int i, count, index, stride;
	float cx, cy, zx, zy, tempx;

	index = blockIdx.x * blockDim.x + threadIdx.x;
	stride = blockDim.x * gridDim.x;
	
	for (i = index; i < xRes * yRes; i += stride) {
		count = 0;
		zx = 0, zy = 0;
		cx = pos[0] + pos[2] * (i % res[0]) / res[0];
		cy = pos[1] + pos[3] * (i / res[0]) / res[1];

		while ((zx * zx + zy * zy < 4) && (count < 255)) {
			tempx = zx * zx - zy * zy + cx;
			zy = 2 * zx * zy + cy;
			zx = tempx;
			++count;
		}

		pixels[i] = 0x00010101 * count;
	}
}

int main(int argc, char** argv) {
	MainWindow();
	return EXIT_SUCCESS;
}

